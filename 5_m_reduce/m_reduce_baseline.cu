#include<hip/hip_runtime.h>

#include<stdio.h>
#include<stdlib.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) {
            exit(code);
        }
    }
}

__global__ void reduce_baseline(float* sum, float *input, size_t n)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    printf("id = %d\n", id);
    // input: global memory
    // sum: global memory
    float s = 0;
    // s: reg
    for(int i = 0; i < n; ++i) 
    {
        s += input[i];
    }
    // reg write into global
    *sum = s;
}


static bool check(float cpu_res, float gpu_res) {
    return cpu_res == gpu_res;
}

// * host端申请内存并初始化数据
// * device端申请内存
// * host端数据拷贝到device端
// * (计时开始)
// * 启动CUDA kernel
// * (计时结束)
// * device端把结果拷贝回host端
// * 检查device端计算结果和host端计算结果
// * 释放host和device端内存

int main()
{
    const int N = 25600000;
    float *da = nullptr, *ha = nullptr;
    float *ds = nullptr;
    // float hs = 0;
        // tips: cudaMemcpy 当host端内存作为dst时, host内存必须位于堆上; 不能位于栈上
    float *hs = nullptr;
    float h_res = 0;
    
    ha = (float*)malloc(N * sizeof(float));         // cpu host mem
    hs = (float*)malloc(sizeof(float));
    gpuErrchk(hipMalloc(&da, N * sizeof(float)));  // gpu global mem
    gpuErrchk(hipMalloc(&ds, sizeof(float)));
    for(int i = 0; i < N; ++i) {
        ha[i] = 1;
        h_res += ha[i];
    }
    // printf("ha %p; da %p.\n", ha, da);
    hipMemcpy(da, ha, N * sizeof(float), hipMemcpyHostToDevice);  
    
    int grid_shape = 1;
    int block_shape = 1;
    hipEvent_t start, stop;
    float milliseconds = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_baseline<<<grid_shape, block_shape>>>(ds, da, N); 
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    // check
    gpuErrchk(hipMemcpy(hs, ds, sizeof(float), hipMemcpyDeviceToHost));
    if (!check(h_res, *hs)) {
        return 0;
    }
    printf("cpu: %.lf, gpu: %.lf\n", h_res, *hs);
    printf("reduce_baseline latency: %.lf ms\n", milliseconds);
    // latency: 562 ms
}