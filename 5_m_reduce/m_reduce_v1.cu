#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<cassert>


// 引入并行 + shared mem
// 并行
    // 每个block之间并行
    // 对于每个block内部的thread, 在不同迭代之间是串行的, 但是在每次迭代内部, thread是并行的.
// shared mem
    // baseline中, 每次加法都是对global mem进行读取, 故我们现在将其转移至shared mem.
    // 每个block都有一个shared mem, 且大小一般为64KB. 所以256 * 4 / 1024 = 1KB, 完全够用.
    // 由于__shared__变量大小要在编译期间确定, 所以模板偏特化来指明__shared__数组大小.
template<int block_size>
__global__ void reduce_v0(float *input, float *output, const int N)     // 这个N必须传值, 不能传常量左值引用！不然会有未定义事情发生！! 找了半天才找到！
{
    // block_size = blockDim.x: 但是blockDim.x是在运行时确定, 所以我们需要通过模板传进来.
    // assert(block_size == blockDim.x); // 256                      
    // shared memory
    __shared__ float sm[block_size];
    // grid 一维; block 一维
    int gid = blockIdx.x * blockDim.x + threadIdx.x;        // 总的thread id. 也是当前线程对应的data在input里的索引
    int tid = threadIdx.x;

    // 保存到shared mem中. 避免迭代的时候反复存取global mem
    if (gid >= N) {
        sm[tid] = 0;
    } else  {
        sm[tid] = input[gid];
    }
    // 等待block内的所有thread对shared mem的load和store完成
    __syncthreads();


    // i: 代表第x轮迭代, 所求和的data的间距为2^x.
    // 最终结果存储到0号位.
    // 这些都是在shared memory上进行计算.

    // before: warp divergence
        // thread tid就对应data[tid]
        // 每轮迭代, 每个thread tid处理两个data: data[tid] 和 data[tid+i]
        // 每轮迭代中, 工作的thread散落在各个warp里面
    // for(int i = 1; i < blockDim.x; i *= 2) {
        // 这样每次都会间隔几个tid, 才会有一个thread进行工作.
    //     if (tid % (i * 2) == 0 && (tid + i) < blockDim.x) {
    //         sm[tid] += sm[tid + i];
    //     }
    //     __syncthreads();        // 应该可以放在分支里面, 因为threadId一次没计算, 剩下的迭代也不会参与. 没进入分支的可以直接return.
    // }

    // after: 消除warp divergence
        // thread tid不对应data[tid], 而是直接对应出要处理的两个data的idx, 并对其进行计算
        // 这样, 每轮迭代中, 工作的thread都紧挨在一起
    // 迭代的还是thread处理数据的间隔
    for (int i = 1; i < blockDim.x; i *= 2) {
        int data_idx = 2 * i * tid;             // 计算出当前tid对应处理的data下标.
        if (data_idx + i < blockDim.x) {
            sm[data_idx] += sm[data_idx + i]; 
        }
        __syncthreads();
    }

    if (tid != 0)
        return ;
    
    // 将结果从shared mem写回global memory
    // 每个block的thread计算出来一个结果, 存入output[blockIdx.x].
    // 最后cpu再把所有block的结果合并到一起
    output[blockIdx.x] = sm[0];

    // printf("block[%d], thread[%d], s = %.lf\n", blockIdx.x, tid, output[blockIdx.x]);
}


bool check(float cpu_res, float *gpu_res, const int n) {
    float gpu_merge = 0;
    for(int i = 0; i < n; ++i) {
        gpu_merge += gpu_res[i];    
    }
    printf("cpu_res == %.lf; gpu_res == %.lf\n", cpu_res, gpu_merge);
    return cpu_res == gpu_merge;    
}

int main()
{
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    const int N = 2560000;
    int nbytes = N * sizeof(float);
    const int block_size = 256;     // const修饰才能在编译时已知
    int grid_size = min((N + block_size - 1) / block_size, deviceProp.maxGridSize[0]);      // min(10000, 2147483647)
    
    float *ha = (float*)malloc(nbytes);
    float *hs = (float*)malloc(grid_size * sizeof(float));
    float h_res = 0;
    for (int i = 0; i < N; ++i) {
        ha[i] = 1;
        h_res += ha[i];
    }

    float *da = nullptr;
    float *ds = nullptr;
    hipMalloc(&da, nbytes);
    hipMalloc(&ds, grid_size * sizeof(float));
    hipMemcpy(da, ha, nbytes, hipMemcpyHostToDevice);

    dim3 grid(grid_size);
    dim3 block(block_size);
    hipEvent_t start, stop;
    float milliseconds = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_v0<block_size><<<grid, block>>>(da, ds, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(hs, ds, grid_size * sizeof(float), hipMemcpyDeviceToHost);
    printf("allcated %d blocks, thread %d per block, data counts are %d\n", grid_size, block_size, N);

    if (!check(h_res, hs, grid_size)) {
        printf("bad ans!\n");
    } else {
        printf("good ans!\n");
        printf("latency %.3lf ms\n", milliseconds);
    }

    // allcated 10000 blocks, thread 256 per block, data counts are 2560000
    // cpu_res == 2560000; gpu_res == 2560000
    // good ans!
    // latency 0.053 ms
    return 0;
}